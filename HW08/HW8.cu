// Name: Kyle Earp
// Vector Dot product on 1 block 
// nvcc HW8.cu -o temp
/*
 What to do:
 This code uses the CPU to compute the dot product of two vectors of length N. 
 It includes a skeleton for setting up a GPU dot product, but that part is currently empty.
 Additionally, the CPU code is somewhat convoluted, but it is structured this way to parallel 
 the GPU code you will need to write. The program will also verify whether you have correctly 
 implemented the dot product on the GPU.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 823 // Length of the vector

// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
float DotCPU, DotGPU;
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;

// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void initialize();
void dotProductCPU(float*, float*, float*, int);
__global__ void dotProductGPU(float*, float*, float*, int);
bool  check(float, float, float);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	BlockSize.x = 1000;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = 1;
	GridSize.y = 1;
	GridSize.z = 1;
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipMalloc(&A_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
}

// Loading values into the vectors that we will multiply.
void initialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(2*i);
	}
}

// Multiplying vectors a and b and storing the scalar result in C_CPU[0].
void dotProductCPU(float *a, float *b, float *C_CPU, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		C_CPU[id] = a[id] * b[id];
	}
	
	for(int id = 1; id < n; id++)
	{ 
		C_CPU[0] += C_CPU[id];
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It multiplies vectors a and b and stores the scalar result in C_GPU[0].
__global__ void dotProductGPU(float *a, float *b, float *C_GPU, int n)
{
	// Get the index of the thread
	int id = threadIdx.x;
	
	// Make sure that the thread is within the range of the vectors.
	if(id < n)
	{
		// Store the product of the elements of a and b in C_GPU.
		C_GPU[id] = a[id] * b[id];
	}

	// Make sure all threads are done before we add up the elements of C_GPU.
	__syncthreads();

	// If the number of threads is odd, then we need to add the last element to the first element and make the number of threads even.
	int blockSize = blockDim.x;
	if(blockSize % 2 == 1)
	{
		blockSize--;
		if(id == 0 && blockSize < n)
		{
			C_GPU[0] += C_GPU[blockSize];
		}
		__syncthreads();
	}

	// Fold the elements of C_GPU into C_GPU[0]. Folding on the size of the block.
	for(int fold = blockSize/2; fold > 0; fold = fold/2)
	{
		// Only the threads that are less than fold will be used to add the elements of C_GPU.
		// id + fold must be inside the array.
		if(id < fold && id + fold < n)
		{
			C_GPU[id] += C_GPU[id + fold];
		}
		// Make sure all threads are done before we move on to the next fold.
		__syncthreads();
		// If fold is greater than 1 and odd, then we need to add the last element to the first element.
		// When the fold is equal to 1, the next iteration takes care of the last element
		if(fold > 1 && fold % 2 == 1)
		{
			fold--;
			if(id == 0)
			{
				C_GPU[0] += C_GPU[fold];
			}
			__syncthreads();
		}
	}
}

// Checking to see if anything went wrong in the vector multiplication.
bool check(float cpuAnswer, float gpuAnswer, float tolerence)
{
	double percentError;
	
	percentError = abs((gpuAnswer - cpuAnswer)/(cpuAnswer))*100.0;
	printf("\n\n percent error = %lf\n", percentError);
	
	if(percentError < Tolerance) 
	{
		return(true);
	}
	else 
	{
		return(false);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void cleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
	
	hipFree(A_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main()
{
	timeval start, end;
	long timeCPU, timeGPU;
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	initialize();
	
	// Adding on the CPU
	gettimeofday(&start, NULL);
	dotProductCPU(A_CPU, B_CPU, C_CPU, N);
	DotCPU = C_CPU[0];
	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	
	// Adding on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	dotProductGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU, C_GPU, N);
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	DotGPU = C_CPU[0]; // C_GPU was copied into C_CPU.
	
	// Making sure the GPU and CPU wiat until each other are at the same place.
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);

	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);
	
	// Checking to see if all went correctly.
	if(check(DotCPU, DotGPU, Tolerance) == false)
	{
		printf("\n\n Something went wrong in the GPU dot product.\n");
	}
	else
	{
		printf("\n\n You did a dot product correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	cleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}


